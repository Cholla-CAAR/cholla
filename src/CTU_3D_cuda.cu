
#include <hip/hip_runtime.h>
/*! \file CTU_3D_cuda.cu
 *  \brief Definitions of the cuda 3D CTU algorithm functions. */

#ifdef CUDA

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"CTU_3D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_ctu_cuda.h"
#include"plmc_ctu_cuda.h"
#include"ppmp_ctu_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"h_correction_3D_cuda.h"
#include"cooling_cuda.h"
#include"subgrid_routines_3D.h"
#include"io.h"

//#define TEST

__global__ void Evolve_Interface_States_3D(Real *dev_conserved, Real *dev_Q_Lx, Real *dev_Q_Rx, Real *dev_F_x,
                                           Real *dev_Q_Ly, Real *dev_Q_Ry, Real *dev_F_y,
                                           Real *dev_Q_Lz, Real *dev_Q_Rz, Real *dev_F_z,
                                           int nx, int ny, int nz, int n_ghost, 
                                           Real dx, Real dy, Real dz, Real dt);


Real CTU_Algorithm_3D_CUDA(Real *host_conserved0, Real *host_conserved1, int nx, int ny, int nz, int x_off, int y_off, int z_off, int n_ghost, Real dx, Real dy, Real dz, Real xbound, Real ybound, Real zbound, Real dt)
{
  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //concatenated into a 1-d array

  int n_fields = 5;
  #ifdef DE
  n_fields++;
  #endif

/*
  #ifdef COOLING_GPU
  // allocate CUDA arrays for cooling/heating tables
  cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc(32, 0, 0, 0, cudaChannelFormatKindFloat);
  cudaArray* cuCoolArray;
  cudaArray* cuHeatArray;
  cudaMallocArray(&cuCoolArray, &channelDesc, 81, 121);
  cudaMallocArray(&cuHeatArray, &channelDesc, 81, 121);
  // Copy to device memory the cooling and heating arrays
  // in host memory
  cudaMemcpyToArray(cuCoolArray, 0, 0, cooling_table, 81*121*sizeof(float), cudaMemcpyHostToDevice);
  cudaMemcpyToArray(cuHeatArray, 0, 0, heating_table, 81*121*sizeof(float), cudaMemcpyHostToDevice);

  // Specify textures
  struct cudaResourceDesc coolResDesc;
  memset(&coolResDesc, 0, sizeof(coolResDesc));
  coolResDesc.resType = cudaResourceTypeArray;
  coolResDesc.res.array.array = cuCoolArray;
  struct cudaResourceDesc heatResDesc;
  memset(&heatResDesc, 0, sizeof(heatResDesc));
  heatResDesc.resType = cudaResourceTypeArray;
  heatResDesc.res.array.array = cuHeatArray;  

  // Specify texture object parameters (same for both tables)
  struct cudaTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = cudaAddressModeClamp; // out-of-bounds fetches return border values
  texDesc.addressMode[1] = cudaAddressModeClamp; // out-of-bounds fetches return border values
  texDesc.filterMode = cudaFilterModeLinear;
  texDesc.readMode = cudaReadModeElementType;
  texDesc.normalizedCoords = 1;

  // Create texture objects
  cudaTextureObject_t coolTexObj = 0;
  cudaCreateTextureObject(&coolTexObj, &coolResDesc, &texDesc, NULL);
  cudaTextureObject_t heatTexObj = 0;
  cudaCreateTextureObject(&heatTexObj, &heatResDesc, &texDesc, NULL);
  #endif
*/

  // number of cells
  int BLOCK_VOL = nx*ny*nz;

  // define the dimensions for the 1D grid
  int  ngrid = (BLOCK_VOL + TPB - 1) / TPB;

  //number of blocks per 1-d grid  
  dim3 dim1dGrid(ngrid, 1, 1);

  //number of threads per 1-d block   
  dim3 dim1dBlock(TPB, 1, 1);


  // St up pointers for the location to copy from and to
  Real *tmp1 = host_conserved0;
  Real *tmp2 = host_conserved1;

  // allocate an array on the CPU to hold max_dti returned from each thread block
  Real max_dti = 0;
  Real *host_dti_array;
  host_dti_array = (Real *) malloc(ngrid*sizeof(Real));

  // allocate GPU arrays
  // conserved variables
  Real *dev_conserved;
  // input states and associated interface fluxes (Q* and F* from Stone, 2008)
  Real *Q_Lx, *Q_Rx, *Q_Ly, *Q_Ry, *Q_Lz, *Q_Rz, *F_x, *F_y, *F_z;
  // arrays to hold the eta values for the H correction
  Real *eta_x, *eta_y, *eta_z, *etah_x, *etah_y, *etah_z;
  // array of inverse timesteps for dt calculation
  Real *dev_dti_array;

#ifdef TEST
  Real *test1, *test2;
  test1 = (Real *) malloc(n_fields*BLOCK_VOL*sizeof(Real));
  test2 = (Real *) malloc(n_fields*BLOCK_VOL*sizeof(Real));
#endif

  // allocate memory on the GPU
  CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Lx,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Rx,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ly,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ry,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Lz,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Rz,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_x,   n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_y,   n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_z,   n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_x,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_y,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_z,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_x, BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_y, BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_z, BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );

  // zero the GPU arrays
  cudaMemset(dev_conserved, 0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Lx,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Rx,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Ly,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Ry,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Lz,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Rz,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(F_x,   0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(F_y,   0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(F_z,   0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(eta_x,  0, BLOCK_VOL*sizeof(Real));
  cudaMemset(eta_y,  0, BLOCK_VOL*sizeof(Real));
  cudaMemset(eta_z,  0, BLOCK_VOL*sizeof(Real));
  cudaMemset(etah_x, 0, BLOCK_VOL*sizeof(Real));
  cudaMemset(etah_y, 0, BLOCK_VOL*sizeof(Real));
  cudaMemset(etah_z, 0, BLOCK_VOL*sizeof(Real));
  cudaMemset(dev_dti_array, 0, ngrid*sizeof(Real));  
  CudaCheckError();


  // copy the conserved variables onto the GPU
  CudaSafeCall( cudaMemcpy(dev_conserved, tmp1, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );
    

  // Step 1: Do the reconstruction
  #ifdef PCM
  PCM_Reconstruction_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, gama);
  #endif //PCM
  #ifdef PLMP
  PLMP_CTU<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0);
  PLMP_CTU<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx, ny, nz, n_ghost, dy, dt, gama, 1);
  PLMP_CTU<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, dz, dt, gama, 2);
  #endif //PLMP 
  #ifdef PLMC
  PLMC_CTU<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0);
  PLMC_CTU<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx, ny, nz, n_ghost, dy, dt, gama, 1);
  PLMC_CTU<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, dz, dt, gama, 2);
  #endif //PLMC 
  #ifdef PPMP
  PPMP_CTU<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0);
  PPMP_CTU<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx, ny, nz, n_ghost, dy, dt, gama, 1);
  PPMP_CTU<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, dz, dt, gama, 2);
  #endif //PPMP
  #ifdef PPMC
  PPMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0);
  PPMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx, ny, nz, n_ghost, dy, dt, gama, 1);
  PPMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, dz, dt, gama, 2);
  #endif //PPMC
  CudaCheckError();


  #ifdef H_CORRECTION
  #ifndef CTU
  calc_eta_x_3D<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, eta_x, nx, ny, nz, n_ghost, gama);
  calc_eta_y_3D<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, eta_y, nx, ny, nz, n_ghost, gama);
  calc_eta_z_3D<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, eta_z, nx, ny, nz, n_ghost, gama);
  CudaCheckError();
  // and etah values for each interface
  calc_etah_x_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_x, nx, ny, nz, n_ghost);
  calc_etah_y_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_y, nx, ny, nz, n_ghost);
  calc_etah_z_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_z, nx, ny, nz, n_ghost);
  CudaCheckError();
  #endif // NO CTU
  #endif // H_CORRECTION


  // Step 2: Calculate the fluxes
  #ifdef EXACT
  Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0);
  Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, 1);
  Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, 2);
  #endif //EXACT
  #ifdef ROE
  Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, etah_x, 0);
  Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, etah_y, 1);
  Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, etah_z, 2);
  #endif //ROE
  #ifdef HLLC
  Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, etah_x, 0);
  Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, etah_y, 1);
  Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, etah_z, 2);
  #endif //HLLC
  CudaCheckError();

#ifdef TEST 
    CudaSafeCall( cudaMemcpy(test1, F_x, 6*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );
    CudaSafeCall( cudaMemcpy(test2, F_y, 6*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );
    for (int i=0; i<nx; i++) {
      for (int j=0; j<ny; j++) {
        int z = n_ghost+8;
        if (test1[i + j*nx + z*nx*ny] != test2[j + i*nx + z*nx*ny]) {
          printf("%3d %3d %f %f\n", i, j, test1[i + j*nx + z*nx*ny], test2[j + i*nx + z*nx*ny]);
        }
      }
    }
#endif

  #ifdef CTU
  // Step 3: Evolve the interface states
  Evolve_Interface_States_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, F_x, Q_Ly, Q_Ry, F_y, Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, dx, dy, dz, dt);
  CudaCheckError();

  #ifdef H_CORRECTION
  // Step 3.5: Calculate eta values for H correction
  calc_eta_x_3D<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, eta_x, nx, ny, nz, n_ghost, gama);
  calc_eta_y_3D<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, eta_y, nx, ny, nz, n_ghost, gama);
  calc_eta_z_3D<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, eta_z, nx, ny, nz, n_ghost, gama);
  CudaCheckError();
  // and etah values for each interface
  calc_etah_x_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_x, nx, ny, nz, n_ghost);
  calc_etah_y_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_y, nx, ny, nz, n_ghost);
  calc_etah_z_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_z, nx, ny, nz, n_ghost);
  CudaCheckError();
  #endif //H_CORRECTION


  // Step 4: Calculate the fluxes again
  #ifdef EXACT
  Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0);
  Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, 1);
  Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, 2);
  #endif //EXACT
  #ifdef ROE
  Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, etah_x, 0);
  Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, etah_y, 1);
  Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, etah_z, 2);
  #endif //ROE
  #ifdef HLLC
  Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, etah_x, 0);
  Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, etah_y, 1);
  Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, etah_z, 2);
  #endif //HLLC
  CudaCheckError();
  #endif //CTU

  // Step 5: Update the conserved variable array
  Update_Conserved_Variables_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, F_x, F_y, F_z, nx, ny, nz, x_off, y_off, z_off, n_ghost, dx, dy, dz, xbound, ybound, zbound, dt, gama);
  CudaCheckError();

  // Synchronize the total and internal energies
  #ifdef DE
  Sync_Energies_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx, ny, nz, n_ghost, gama);
  CudaCheckError();
  #endif

  // Apply cooling
  #ifdef COOLING_GPU
  //cooling_kernel<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx, ny, nz, n_ghost, dt, gama, coolTexObj, heatTexObj);
  cooling_kernel<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx, ny, nz, n_ghost, dt, gama);
  CudaCheckError();
  #endif

  // Step 6: Calculate the next timestep
  Calc_dt_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx, ny, nz, n_ghost, dx, dy, dz, dev_dti_array, gama);
  CudaCheckError();

  // copy the updated conserved variable array back to the CPU
  CudaSafeCall( cudaMemcpy(tmp2, dev_conserved, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );
  CudaCheckError();

  // copy the dti array onto the CPU
  CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
  // iterate through to find the maximum inverse dt for this subgrid block
  for (int i=0; i<ngrid; i++) {
    max_dti = fmax(max_dti, host_dti_array[i]);
  }


  // free CPU memory
  free(host_dti_array);  

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(Q_Ly);
  cudaFree(Q_Ry);
  cudaFree(Q_Lz);
  cudaFree(Q_Rz);
  cudaFree(F_x);
  cudaFree(F_y);
  cudaFree(F_z);
  cudaFree(eta_x);
  cudaFree(eta_y);
  cudaFree(eta_z);
  cudaFree(etah_x);
  cudaFree(etah_y);
  cudaFree(etah_z);
  cudaFree(dev_dti_array);
  #ifdef COOLING_GPU
  // Destroy texture object
  //cudaDestroyTextureObject(coolTexObj);
  //cudaDestroyTextureObject(heatTexObj);
  // Free device memory
  //cudaFreeArray(cuCoolArray);
  //cudaFreeArray(cuHeatArray);  
  #endif

#ifdef TEST
  free(test1);
  free(test2);
#endif
 
  // return the maximum inverse timestep
  return max_dti;

}


__global__ void Evolve_Interface_States_3D(Real *dev_conserved, Real *dev_Q_Lx, Real *dev_Q_Rx, Real *dev_F_x,
                                           Real *dev_Q_Ly, Real *dev_Q_Ry, Real *dev_F_y,
                                           Real *dev_Q_Lz, Real *dev_Q_Rz, Real *dev_F_z,
                                           int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real dt)
{
  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  int n_cells = nx*ny*nz;

  // get a thread ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int zid = tid / (nx*ny);
  int yid = (tid - zid*nx*ny) / nx;
  int xid = tid - zid*nx*ny - yid*nx;
  int id = xid + yid*nx + zid*nx*ny;

  if (xid > n_ghost-3 && xid < nx-n_ghost+1 && yid > n_ghost-2 && yid < ny-n_ghost+1 && zid > n_ghost-2 && zid < nz-n_ghost+1)
  {
    // set the new x interface states
    // left
    int ipo = xid+1 + yid*nx + zid*nx*ny;
    int jmo = xid + (yid-1)*nx + zid*nx*ny;
    int kmo = xid + yid*nx + (zid-1)*nx*ny;
    int ipojmo = xid+1 + (yid-1)*nx + zid*nx*ny;
    int ipokmo = xid+1 + yid*nx + (zid-1)*nx*ny;
    dev_Q_Lx[            id] += 0.5*dtody*(dev_F_y[            jmo] - dev_F_y[            id])
                              + 0.5*dtodz*(dev_F_z[            kmo] - dev_F_z[            id]);
    dev_Q_Lx[  n_cells + id] += 0.5*dtody*(dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id])
                              + 0.5*dtodz*(dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id]);
    dev_Q_Lx[2*n_cells + id] += 0.5*dtody*(dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id])
                              + 0.5*dtodz*(dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id]);
    dev_Q_Lx[3*n_cells + id] += 0.5*dtody*(dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id])
                              + 0.5*dtodz*(dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id]);
    dev_Q_Lx[4*n_cells + id] += 0.5*dtody*(dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id])
                              + 0.5*dtodz*(dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id]);
    #ifdef DE
    dev_Q_Lx[5*n_cells + id] += 0.5*dtody*(dev_F_y[5*n_cells + jmo] - dev_F_y[5*n_cells + id])
                              + 0.5*dtodz*(dev_F_z[5*n_cells + kmo] - dev_F_z[5*n_cells + id]);
    #endif

    // right
    dev_Q_Rx[            id] += 0.5*dtody*(dev_F_y[            ipojmo] - dev_F_y[            ipo])
                              + 0.5*dtodz*(dev_F_z[            ipokmo] - dev_F_z[            ipo]); 
    dev_Q_Rx[  n_cells + id] += 0.5*dtody*(dev_F_y[  n_cells + ipojmo] - dev_F_y[  n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[  n_cells + ipokmo] - dev_F_z[  n_cells + ipo]);
    dev_Q_Rx[2*n_cells + id] += 0.5*dtody*(dev_F_y[2*n_cells + ipojmo] - dev_F_y[2*n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[2*n_cells + ipokmo] - dev_F_z[2*n_cells + ipo]);
    dev_Q_Rx[3*n_cells + id] += 0.5*dtody*(dev_F_y[3*n_cells + ipojmo] - dev_F_y[3*n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[3*n_cells + ipokmo] - dev_F_z[3*n_cells + ipo]);
    dev_Q_Rx[4*n_cells + id] += 0.5*dtody*(dev_F_y[4*n_cells + ipojmo] - dev_F_y[4*n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[4*n_cells + ipokmo] - dev_F_z[4*n_cells + ipo]);
    #ifdef DE
    dev_Q_Rx[5*n_cells + id] += 0.5*dtody*(dev_F_y[5*n_cells + ipojmo] - dev_F_y[5*n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[5*n_cells + ipokmo] - dev_F_z[5*n_cells + ipo]);
    #endif
  }
  if (yid > n_ghost-3 && yid < ny-n_ghost+1 && xid > n_ghost-2 && xid < nx-n_ghost+1 && zid > n_ghost-2 && zid < nz-n_ghost+1)
  {
    // set the new y interface states
    // left
    int jpo = xid + (yid+1)*nx + zid*nx*ny;
    int imo = xid-1 + yid*nx + zid*nx*ny;
    int kmo = xid + yid*nx + (zid-1)*nx*ny;
    int jpoimo = xid-1 + (yid+1)*nx + zid*nx*ny;
    int jpokmo = xid + (yid+1)*nx + (zid-1)*nx*ny;
    dev_Q_Ly[            id] += 0.5*dtodz*(dev_F_z[            kmo] - dev_F_z[            id])
                              + 0.5*dtodx*(dev_F_x[            imo] - dev_F_x[            id]);
    dev_Q_Ly[  n_cells + id] += 0.5*dtodz*(dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id])
                              + 0.5*dtodx*(dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id]);
    dev_Q_Ly[2*n_cells + id] += 0.5*dtodz*(dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id])
                              + 0.5*dtodx*(dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id]);
    dev_Q_Ly[3*n_cells + id] += 0.5*dtodz*(dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id])
                              + 0.5*dtodx*(dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id]);
    dev_Q_Ly[4*n_cells + id] += 0.5*dtodz*(dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id])
                              + 0.5*dtodx*(dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id]);
    #ifdef DE
    dev_Q_Ly[5*n_cells + id] += 0.5*dtodz*(dev_F_z[5*n_cells + kmo] - dev_F_z[5*n_cells + id])
                              + 0.5*dtodx*(dev_F_x[5*n_cells + imo] - dev_F_x[5*n_cells + id]);
    #endif

    // right
    dev_Q_Ry[            id] += 0.5*dtodz*(dev_F_z[            jpokmo] - dev_F_z[            jpo])
                              + 0.5*dtodx*(dev_F_x[            jpoimo] - dev_F_x[            jpo]); 
    dev_Q_Ry[  n_cells + id] += 0.5*dtodz*(dev_F_z[  n_cells + jpokmo] - dev_F_z[  n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[  n_cells + jpoimo] - dev_F_x[  n_cells + jpo]);
    dev_Q_Ry[2*n_cells + id] += 0.5*dtodz*(dev_F_z[2*n_cells + jpokmo] - dev_F_z[2*n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[2*n_cells + jpoimo] - dev_F_x[2*n_cells + jpo]);
    dev_Q_Ry[3*n_cells + id] += 0.5*dtodz*(dev_F_z[3*n_cells + jpokmo] - dev_F_z[3*n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[3*n_cells + jpoimo] - dev_F_x[3*n_cells + jpo]);
    dev_Q_Ry[4*n_cells + id] += 0.5*dtodz*(dev_F_z[4*n_cells + jpokmo] - dev_F_z[4*n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[4*n_cells + jpoimo] - dev_F_x[4*n_cells + jpo]);    
    #ifdef DE
    dev_Q_Ry[5*n_cells + id] += 0.5*dtodz*(dev_F_z[5*n_cells + jpokmo] - dev_F_z[5*n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[5*n_cells + jpoimo] - dev_F_x[5*n_cells + jpo]);    
    #endif
  }
  if (zid > n_ghost-3 && zid < nz-n_ghost+1 && xid > n_ghost-2 && xid < nx-n_ghost+1 && yid > n_ghost-2 && yid < ny-n_ghost+1)
  {
    // set the new z interface states
    // left
    int kpo = xid + yid*nx + (zid+1)*nx*ny;
    int imo = xid-1 + yid*nx + zid*nx*ny;
    int jmo = xid + (yid-1)*nx + zid*nx*ny;
    int kpoimo = xid-1 + yid*nx + (zid+1)*nx*ny;
    int kpojmo = xid + (yid-1)*nx + (zid+1)*nx*ny;
    dev_Q_Lz[            id] += 0.5*dtodx*(dev_F_x[            imo] - dev_F_x[            id])
                              + 0.5*dtody*(dev_F_y[            jmo] - dev_F_y[            id]);
    dev_Q_Lz[  n_cells + id] += 0.5*dtodx*(dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id])
                              + 0.5*dtody*(dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id]);
    dev_Q_Lz[2*n_cells + id] += 0.5*dtodx*(dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id])
                              + 0.5*dtody*(dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id]);
    dev_Q_Lz[3*n_cells + id] += 0.5*dtodx*(dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                              + 0.5*dtody*(dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id]);
    dev_Q_Lz[4*n_cells + id] += 0.5*dtodx*(dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                              + 0.5*dtody*(dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id]);
    #ifdef DE
    dev_Q_Lz[5*n_cells + id] += 0.5*dtodx*(dev_F_x[5*n_cells + imo] - dev_F_x[5*n_cells + id])
                              + 0.5*dtody*(dev_F_y[5*n_cells + jmo] - dev_F_y[5*n_cells + id]);
    #endif
    // right
    dev_Q_Rz[            id] += 0.5*dtodx*(dev_F_x[            kpoimo] - dev_F_x[            kpo])
                              + 0.5*dtody*(dev_F_y[            kpojmo] - dev_F_y[            kpo]); 
    dev_Q_Rz[  n_cells + id] += 0.5*dtodx*(dev_F_x[  n_cells + kpoimo] - dev_F_x[  n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[  n_cells + kpojmo] - dev_F_y[  n_cells + kpo]);
    dev_Q_Rz[2*n_cells + id] += 0.5*dtodx*(dev_F_x[2*n_cells + kpoimo] - dev_F_x[2*n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[2*n_cells + kpojmo] - dev_F_y[2*n_cells + kpo]);
    dev_Q_Rz[3*n_cells + id] += 0.5*dtodx*(dev_F_x[3*n_cells + kpoimo] - dev_F_x[3*n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[3*n_cells + kpojmo] - dev_F_y[3*n_cells + kpo]);
    dev_Q_Rz[4*n_cells + id] += 0.5*dtodx*(dev_F_x[4*n_cells + kpoimo] - dev_F_x[4*n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[4*n_cells + kpojmo] - dev_F_y[4*n_cells + kpo]);    
    #ifdef DE
    dev_Q_Rz[5*n_cells + id] += 0.5*dtodx*(dev_F_x[5*n_cells + kpoimo] - dev_F_x[5*n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[5*n_cells + kpojmo] - dev_F_y[5*n_cells + kpo]);    
    #endif
  }

}



#endif //CUDA
