
#include <hip/hip_runtime.h>
/*! \file VL_2D_cuda.cu
 *  \brief Definitions of the cuda 2D VL algorithm functions. */

#ifdef CUDA
#ifdef VL

#include<stdio.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"VL_2D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_vl_cuda.h"
#include"ppmp_vl_cuda.h"
#include"ppmc_vl_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"h_correction_2D_cuda.h"
#include"cooling.h"
#include"subgrid_routines_2D.h"



__global__ void Update_Conserved_Variables_2D_notime(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F_x, Real *dev_F_y, int nx, int ny,
                                              int n_ghost, Real dx, Real dy, Real dt, Real gamma);

__global__ void Update_Conserved_Variables_2D_wtime(Real *dev_conserved, Real *dev_F_x, Real *dev_F_y, int nx, int ny,
                                              int n_ghost, Real dx, Real dy, Real dt, Real *dti_array, Real gamma);


Real VL_Algorithm_2D_CUDA(Real *host_conserved, int nx, int ny, int n_ghost, Real dx, Real dy, Real dt)
{

  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //concatenated into a 1-d array

  #ifdef TIME
  // capture the start time
  cudaEvent_t start, stop;
  cudaEventCreate(&start);
  cudaEventCreate(&stop);
  float elapsedTime;
  #endif

  #ifdef DE
  printf("Dual energy not supported in Van Leer integrator. Use CTU, or neither.\n");
  exit(0);
  #endif
  #ifndef DE
  int n_fields = 5;
  #endif

  // dimensions of subgrid blocks
  int nx_s; //number of cells in the subgrid block along x direction
  int ny_s; //number of cells in the subgrid block along y direction
  int nz_s = 1; //number of cells in the subgrid block along z direction

  // total number of blocks needed
  int block_tot;    //total number of subgrid blocks (unsplit == 1)
  int block1_tot;   //total number of subgrid blocks in x direction
  int block2_tot;   //total number of subgrid blocks in y direction
  int remainder1;   //modulus of number of cells after block subdivision in x direction
  int remainder2;   //modulus of number of cells after block subdivision in y direction 

  // counter for which block we're on
  int block = 0;

  // calculate the dimensions for each subgrid block
  sub_dimensions_2D(nx, ny, n_ghost, &nx_s, &ny_s, &block1_tot, &block2_tot, &remainder1, &remainder2, n_fields);
  //printf("%d %d %d %d %d %d\n", nx_s, ny_s, block1_tot, block2_tot, remainder1, remainder2);
  block_tot = block1_tot*block2_tot;

  // number of cells in one subgrid block
  int BLOCK_VOL = nx_s*ny_s*nz_s;

  // define the dimensions for the 2D grid
  int  ngrid = (BLOCK_VOL + 2*TPB - 1) / (2*TPB);

  //number of blocks per 2-d grid  
  dim3 dim2dGrid(ngrid, 2, 1);

  //number of threads per 1-d block   
  dim3 dim1dBlock(TPB, 1, 1);

  // allocate buffer arrays to copy conserved variable slices into
  Real **buffer;
  allocate_buffers_2D(block1_tot, block2_tot, BLOCK_VOL, buffer, n_fields);
  // and set up pointers for the location to copy from and to
  Real *tmp1;
  Real *tmp2;

  // allocate an array on the CPU to hold max_dti returned from each thread block
  Real max_dti = 0;
  Real *host_dti_array;
  host_dti_array = (Real *) malloc(2*ngrid*sizeof(Real));

  // allocate GPU arrays
  // conserved variables
  Real *dev_conserved, *dev_conserved_half;
  // input states and associated interface fluxes (Q* and F* from Stone, 2008)
  Real *Q_Lx, *Q_Rx, *Q_Ly, *Q_Ry, *F_x, *F_y;
  // arrays to hold the eta values for the H correction
  Real *eta_x, *eta_y, *etah_x, *etah_y;
  // array of inverse timesteps for dt calculation
  Real *dev_dti_array;


  // allocate memory on the GPU
  CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_conserved_half, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Lx, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Rx, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ly, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ry, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_x,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_y,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_x,   BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_y,   BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_x,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_y,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_dti_array, 2*ngrid*sizeof(Real)) );
  

  // transfer first conserved variable slice into the first buffer
  host_copy_init_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, remainder1, BLOCK_VOL, host_conserved, buffer, &tmp1, &tmp2, n_fields);
  
  // START LOOP OVER SUBGRID BLOCKS HERE
  while (block < block_tot) {

    // zero all the GPU arrays
    cudaMemset(dev_conserved, 0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(dev_conserved_half, 0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Lx,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Rx,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Ly,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Ry,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(F_x,   0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(F_y,   0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(eta_x,  0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(eta_y,  0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(etah_x, 0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(etah_y, 0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(dev_dti_array, 0, 2*ngrid*sizeof(Real));
    CudaCheckError();

    // copy the conserved variables onto the GPU
    CudaSafeCall( cudaMemcpy(dev_conserved, tmp1, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );


    // Step 1: Use PCM reconstruction to put conserved variables into interface arrays
    PCM_Reconstruction_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, nx_s, ny_s, n_ghost, gama);
    CudaCheckError();

    // Step 2: Calculate first-order upwind fluxes 
    #ifdef EXACT
    Calculate_Exact_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0);
    Calculate_Exact_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1);
    #endif
    #ifdef ROE
    Calculate_Roe_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0);
    Calculate_Roe_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1);
    #endif
    CudaCheckError();


    // Step 3: Update the conserved variables half a timestep 
    Update_Conserved_Variables_2D_notime<<<dim2dGrid,dim1dBlock>>>(dev_conserved, dev_conserved_half, F_x, F_y, nx_s, ny_s, n_ghost, dx, dy, 0.5*dt, gama);
    CudaCheckError();


    // Step 4: Construct left and right interface values using updated conserved variables
    #ifdef PLMP
    PLMP_VL<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, gama, 0);
    PLMP_VL<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, gama, 1);
    #endif
    #ifdef PLMC
    printf("PLMC not supported for Van Leer integrator.\n");
    #endif
    #ifdef PPMP
    PPMP_VL<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, gama, 0);
    PPMP_VL<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, gama, 1);
    #endif //PPMP
    #ifdef PPMC
    PPMC_VL<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, gama, 0);
    PPMC_VL<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, gama, 1);
    #endif //PPMC
    CudaCheckError();


    #ifdef H_CORRECTION
    // Step 4.5: Calculate eta values for H correction
    calc_eta_x_2D<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, eta_x, nx_s, ny_s, n_ghost, gama);
    calc_eta_y_2D<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, eta_y, nx_s, ny_s, n_ghost, gama);
    CudaCheckError();
    // and etah values for each interface
    calc_etah_x_2D<<<dim2dGrid,dim1dBlock>>>(eta_x, eta_y, etah_x, nx_s, ny_s, n_ghost);
    calc_etah_y_2D<<<dim2dGrid,dim1dBlock>>>(eta_x, eta_y, etah_y, nx_s, ny_s, n_ghost);
    CudaCheckError();
    #endif


    // Step 5: Calculate the fluxes again
    #ifdef EXACT
    Calculate_Exact_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0);
    Calculate_Exact_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1);
    #endif
    #ifdef ROE
    Calculate_Roe_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0);
    Calculate_Roe_Fluxes<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1);
    #endif
    CudaCheckError();


    // Step 5: Update the conserved variable array
    Update_Conserved_Variables_2D_wtime<<<dim2dGrid,dim1dBlock>>>(dev_conserved, F_x, F_y, nx_s, ny_s, n_ghost, dx, dy, dt, dev_dti_array, gama);
    CudaCheckError();


    // Apply cooling
    #ifdef COOLING_GPU
    cooling_kernel<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, dt, gama);
    #endif


    // copy the conserved variable array back to the CPU
    CudaSafeCall( cudaMemcpy(tmp2, dev_conserved, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );

    // copy the next conserved variable blocks into appropriate buffers
    host_copy_next_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, block2_tot, remainder1, remainder2, BLOCK_VOL, host_conserved, buffer, &tmp1, n_fields);

    // copy the updated conserved variable array back into the host_conserved array on the CPU
    host_return_values_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, block2_tot, remainder1, remainder2, BLOCK_VOL, host_conserved, buffer, n_fields);


    // copy the dti array onto the CPU
    CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, 2*ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // iterate through to find the maximum inverse dt for this subgrid block
    for (int i=0; i<2*ngrid; i++) {
      max_dti = fmax(max_dti, host_dti_array[i]);
    }


    // add one to the counter
    block++;

  }


  // free the CPU memory
  free(host_dti_array);
  free_buffers_2D(nx, ny, nx_s, ny_s, block1_tot, block2_tot, buffer);

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(dev_conserved_half);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(Q_Ly);
  cudaFree(Q_Ry);
  cudaFree(F_x);
  cudaFree(F_y);
  cudaFree(eta_x);
  cudaFree(eta_y);
  cudaFree(etah_x);
  cudaFree(etah_y);
  cudaFree(dev_dti_array);


  // return the maximum inverse timestep
  return max_dti;

}


__global__ void Update_Conserved_Variables_2D_notime(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F_x, Real *dev_F_y, int nx, int ny, int n_ghost, Real dx, Real dy, Real dt, Real gamma)
{
  int id, xid, yid, n_cells;
  int imo, jmo;

  Real dtodx = dt/dx;
  Real dtody = dt/dy;

  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;


  // all threads but one outer ring of ghost cells 
  if (xid > 0 && xid < nx-1 && yid > 0 && yid < ny-1)
  {
    // update the conserved variable array
    imo = xid-1 + yid*nx;
    jmo = xid + (yid-1)*nx;
    dev_conserved_half[            id] = dev_conserved[            id] 
                                       + dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                       + dtody * (dev_F_y[            jmo] - dev_F_y[            id]);
    dev_conserved_half[  n_cells + id] = dev_conserved[  n_cells + id] 
                                       + dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id]) 
                                       + dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id]);
    dev_conserved_half[2*n_cells + id] = dev_conserved[2*n_cells + id] 
                                       + dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id]) 
                                       + dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id]); 
    dev_conserved_half[3*n_cells + id] = dev_conserved[3*n_cells + id] 
                                       + dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                       + dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id]);
    dev_conserved_half[4*n_cells + id] = dev_conserved[4*n_cells + id] 
                                       + dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                       + dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id]);
  } 
}



__global__ void Update_Conserved_Variables_2D_wtime(Real *dev_conserved, Real *dev_F_x, Real *dev_F_y, int nx, int ny, int n_ghost, Real dx, Real dy, Real dt, Real *dti_array, Real gamma)
{
  __shared__ Real max_dti[TPB];

  Real d, d_inv, vx, vy, vz, P, cs;
  int id, tid, xid, yid, n_cells;
  int imo, jmo;

  Real dtodx = dt/dx;
  Real dtody = dt/dy;

  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;
  // and a thread id within the block
  tid = threadIdx.x;

  // set shared memory to 0
  max_dti[tid] = 0;
  __syncthreads();

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    // update the conserved variable array
    imo = xid-1 + yid*nx;
    jmo = xid + (yid-1)*nx;
    dev_conserved[            id] += dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                  +  dtody * (dev_F_y[            jmo] - dev_F_y[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id]) 
                                  +  dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id]) 
                                  +  dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id]); 
    dev_conserved[3*n_cells + id] += dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                  +  dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                  +  dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id]);
   

    // start timestep calculation here
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    P  = fmax(P, (Real) 1.0e-20);
    // find the max wavespeed in that cell, use it to calculate the inverse timestep
    cs = sqrt(d_inv * gamma * P);
    max_dti[tid] = fmax((fabs(vx)+cs)/dx, (fabs(vy)+cs)/dy);
  }
  __syncthreads();
  
  // do the reduction in shared memory (find the max inverse timestep in the block)
  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      max_dti[tid] = fmax(max_dti[tid], max_dti[tid + s]);
    }
    __syncthreads();
  }

  // write the result for this block to global memory
  if (tid == 0) dti_array[blockId] = max_dti[0];
  
}

#endif //VL
#endif //CUDA

